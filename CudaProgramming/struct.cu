#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include"struct.hpp"
#include<cstdio>
#include<string>


__global__ void __test(ExampleStruct *exampleStruct) {
	printf("Device: %d %.3f %s\n", exampleStruct->a, exampleStruct->b, exampleStruct->c);
}

void TestStruct() {
	ExampleStruct *hostExampleStruct = new ExampleStruct();
	ExampleStruct *deviceExampleStruct;
	char *text;
	hipMalloc(&deviceExampleStruct, sizeof(ExampleStruct));
	hipMalloc(&text, strlen(hostExampleStruct->c));
	hipMemcpy(
		deviceExampleStruct, hostExampleStruct, sizeof(ExampleStruct), hipMemcpyHostToDevice
	);
	hipMemcpy(text, hostExampleStruct->c, strlen(hostExampleStruct->c), hipMemcpyHostToDevice);
	hipMemcpy(&(deviceExampleStruct->c), &text, sizeof(text), hipMemcpyHostToDevice);

	printf("  Host: %d %.3f %s\n", hostExampleStruct->a, hostExampleStruct->b, hostExampleStruct->c);
	__test << <1, 1 >> > (deviceExampleStruct);

	hipFree(deviceExampleStruct);
	delete hostExampleStruct;
}