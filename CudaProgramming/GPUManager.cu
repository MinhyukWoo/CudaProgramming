#include "hip/hip_runtime.h"
#include"GPUManager.cuh"
#include ""
#include<stdio.h>


__global__ void Init() {

}

GPUManager::GPUManager() : _threadSize(256) {
	Init << <1, 1 >> > ();
};

__global__ void ProcessOne1D(WORD *deviceDstPtr, WORD *deviceSrcPtr, size_t size, kernelOne1D_t kernelOne1D) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		kernelOne1D(deviceDstPtr, deviceSrcPtr, tid);
	}
}

void GPUManager::Process(WORD *dstPtr, WORD *srcPtr, size_t size, kernelOne1D_t kernelOne1D) {
	WORD *deviceSrcPtr, *deviceDstPtr;
	hipMalloc(&deviceSrcPtr, sizeof(WORD) * size);
	hipMemcpy(deviceSrcPtr, srcPtr, sizeof(WORD) * size, hipMemcpyHostToDevice);
	hipMalloc(&deviceDstPtr, sizeof(WORD) * size);
	ProcessOne1D << <1 + size / _threadSize, _threadSize >> > (deviceDstPtr, deviceSrcPtr, size, kernelOne1D);
	hipMemcpy(dstPtr, deviceDstPtr, sizeof(WORD) * size, hipMemcpyDeviceToHost);
	hipFree(deviceSrcPtr);
	hipFree(deviceDstPtr);
}


__global__ void ProcessTwo1D(WORD *deviceDstPtr, WORD *deviceSrcPtr1, WORD *deviceSrcPtr2, size_t size, kernelTwo1D_t kernelTwo1D) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		(*kernelTwo1D)(deviceDstPtr, deviceSrcPtr1, deviceSrcPtr2, tid);
	}
}


__device__ void addTwoWords_(WORD *dstPtr, WORD *srcPtr1, WORD *srcPtr2, size_t index) {
	dstPtr[index] = srcPtr1[index] < USHRT_MAX - srcPtr2[index] ? srcPtr1[index] + srcPtr2[index] : USHRT_MAX;
}

__device__ kernelTwo1D_t addTwoWords = addTwoWords_;


void GPUManager::Process(WORD *dstPtr,WORD *srcPtr1, WORD *srcPtr2, size_t size, kernelTwo1D_t kernelTwo1D) {
	WORD *deviceDstPtr, *deviceSrcPtr1, *deviceSrcPtr2;
	hipMalloc(&deviceSrcPtr1, sizeof(WORD) * size);
	hipMemcpy(deviceSrcPtr1, srcPtr1, sizeof(WORD) * size, hipMemcpyHostToDevice);
	hipMalloc(&deviceSrcPtr2, sizeof(WORD) * size);
	hipMemcpy(deviceSrcPtr2, srcPtr2, sizeof(WORD) * size, hipMemcpyHostToDevice);
	hipMalloc(&deviceDstPtr, sizeof(WORD) * size);
	
	
	kernelTwo1D_t hostKernelTwo1D;
	hipMemcpyFromSymbol(&hostKernelTwo1D, HIP_SYMBOL(addTwoWords), sizeof(addTwoWords));
	
	ProcessTwo1D << <1 + size / _threadSize, _threadSize >> > (deviceDstPtr, deviceSrcPtr1, deviceSrcPtr2, size, hostKernelTwo1D);
	hipDeviceSynchronize();
	hipMemcpy(dstPtr, deviceDstPtr, sizeof(WORD) * size, hipMemcpyDeviceToHost);
	hipFree(deviceSrcPtr1);
	hipFree(deviceSrcPtr2);
	hipFree(deviceDstPtr);
}

