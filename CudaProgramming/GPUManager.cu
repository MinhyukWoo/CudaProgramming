#include "hip/hip_runtime.h"
#pragma once
#include "GPUManager.cuh"
#include "GPUTypes.cuh"
#include "GPUEnum.cuh"
#include "GPUFunctions.cuh"
#include ""
#include <stdio.h>


__global__ void Init() {

}

GPUManager::GPUManager() : _threadSize(256) {
	Init << <1, 1 >> > ();
};

__global__ void ProcessTwo1D(WORD *deviceDstPtr, WORD *deviceSrcPtr1, WORD *deviceSrcPtr2, size_t size, kernelTwo1D_t kernelTwo1D) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		(*kernelTwo1D)(deviceDstPtr, deviceSrcPtr1, deviceSrcPtr2, tid);
	}
}


void GPUManager::Process(WORD *dstPtr,WORD *srcPtr1, WORD *srcPtr2, size_t size, E_CUDA_FUNC indexKernelsTwo1D) {
	WORD *deviceDstPtr, *deviceSrcPtr1, *deviceSrcPtr2;
	hipMalloc(&deviceSrcPtr1, sizeof(WORD) * size);
	hipMemcpy(deviceSrcPtr1, srcPtr1, sizeof(WORD) * size, hipMemcpyHostToDevice);
	hipMalloc(&deviceSrcPtr2, sizeof(WORD) * size);
	hipMemcpy(deviceSrcPtr2, srcPtr2, sizeof(WORD) * size, hipMemcpyHostToDevice);
	hipMalloc(&deviceDstPtr, sizeof(WORD) * size);
	
	kernelTwo1D_t hostKernelTwo1D;
	auto error = hipMemcpyFromSymbol(
		&hostKernelTwo1D, HIP_SYMBOL(kernelsTwo1D[0]), sizeof(kernelsTwo1D[0])
	);
	printf("%s\n", hipGetErrorString(error));
	
	ProcessTwo1D << <1 + size / _threadSize, _threadSize >> > (deviceDstPtr, deviceSrcPtr1, deviceSrcPtr2, size, hostKernelTwo1D);
	hipDeviceSynchronize();
	hipMemcpy(dstPtr, deviceDstPtr, sizeof(WORD) * size, hipMemcpyDeviceToHost);
	hipFree(deviceSrcPtr1);
	hipFree(deviceSrcPtr2);
	hipFree(deviceDstPtr);
}

