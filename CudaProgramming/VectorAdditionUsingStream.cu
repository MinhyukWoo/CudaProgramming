#include "hip/hip_runtime.h"
#include "VectorAdditionUsingStream.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void set_element_random(int* devicePtr, hiprandState* deviceStates, int lengthData, int seed) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < lengthData) {
		hiprand_init(seed, idx, 0, &deviceStates[idx]);
		devicePtr[idx] = hiprand(&deviceStates[idx]) % 100 + 1;
	}
}

__global__ void tmp() {

}

VectorAdditionUsingStream::VectorAdditionUsingStream(int lengthData) {
	tmp << <1, 1 >> > ();
	time_t startTime, memcpyTime, synchronizeTime, endTime;
	startTime = clock();
	_lengthData = lengthData;
	int* devicePtr;
	hiprandState * deviceStates;
	int seed = (int)time(NULL);
	
	hipHostMalloc(&_data, sizeof(int) * lengthData);
	hipMalloc(&devicePtr, sizeof(int) * lengthData);
	hipMalloc(&deviceStates, sizeof(hiprandState) * lengthData);
	
	set_element_random << <1 + lengthData / 256, 256, 0 >> > (devicePtr, deviceStates, lengthData, seed);
	hipMemcpy(_data, devicePtr, sizeof(int) * lengthData, hipMemcpyDeviceToHost);
	memcpyTime = clock();

	hipDeviceSynchronize();
	synchronizeTime = clock();

	hipFree(&devicePtr);
	hipFree(&deviceStates);
	endTime = clock();
	std::cout << "Default Stream���� hipMemcpy���� �ɸ� �ð� : " << memcpyTime - startTime << std::endl;
	std::cout << "Device Synchronzie���� �ɸ��� �ð� : " << synchronizeTime - startTime<< std::endl;
	std::cout << "�ҿ� �ð� : " << endTime - startTime << std::endl;
}

VectorAdditionUsingStream::VectorAdditionUsingStream(int lengthData, int lengthStream) {
	time_t startTime;
	startTime = clock();
	_lengthData = lengthData;
	int* devicePtr;
	hiprandState * deviceStates;
	int seed = (int)time(NULL);

	hipHostMalloc(&_data, sizeof(int) * lengthData);
	hipMalloc(&devicePtr, sizeof(int) * lengthData);
	hipMalloc(&deviceStates, sizeof(hiprandState) * lengthData);

	int subLength = lengthData / lengthStream;
	int rest = lengthData % subLength;
	int currentPosition = 0;
	hipStream_t *streams = new hipStream_t[lengthStream];
	std::cout << "�� Stream �� hipMemcpy���� �ɸ� �ð�" << std::endl;
	for (int i = 0; i < lengthStream; i++) {
		hipStreamCreate(&streams[i]);
		int currentLength = subLength;
		if (i == lengthStream-1) {
			currentLength += rest;
		}
		set_element_random << <1 + lengthData / 256, 256, 0, streams[i] >> > (devicePtr + currentPosition, deviceStates, currentLength, seed);
		hipMemcpyAsync(_data + currentPosition, devicePtr + currentPosition, sizeof(int) * currentLength, hipMemcpyDeviceToHost, streams[i]);
		std::cout << i << ":" << clock() - startTime << std::endl;
		currentPosition += subLength;
	}
	std::cout << "�� Stream �� Synchronzie���� �ɸ��� �ð�" << std::endl;
	for (int i = 0; i < lengthStream; i++) {
		hipStreamSynchronize(streams[i]);
		hipStreamDestroy(streams[i]);
		std::cout << i << ":" << clock() - startTime << std::endl;
	}

	hipFree(&devicePtr);
	hipFree(&deviceStates);
	time_t endTime;
	endTime = clock();
	std::cout << "�ҿ� �ð�" << endTime - startTime << std::endl;
}


VectorAdditionUsingStream::~VectorAdditionUsingStream() {

}

void VectorAdditionUsingStream::Process() {
	
}

void VectorAdditionUsingStream::Print() {
	std::cout << "Array:" << std::endl;
	for (int i = 0; i < _lengthData; i++) {
		std::cout << _data[i] << " ";
	}
	std::cout << std::endl;
}