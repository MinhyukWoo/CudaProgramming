#include "hip/hip_runtime.h"
#include"Reduce.hpp"
#include <hiprand/hiprand_kernel.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include<hip/hip_cooperative_groups.h>
#include<cooperative_groups/reduce.h>

__global__ void BadSum(int* dst, int *src, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		int prevDst = *dst;
		*dst += src[tid];
		printf("%2d: %2d + %2d = %2d\n", tid, src[tid], prevDst, *dst);
	}
}



int GetBadSum(int * ptr, int size) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);

	BadSum << <1, 256 >> > (deviceDst, deviceSrc, size);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);

	return hostDst;
}

typedef void (*BinaryOperation)(int *, const int&, const int&);

__device__ void Plus(int * result, const int& lhs, const int& rhs) {
	*result = lhs + rhs;
}

__device__ void Max(int * result, const int& lhs, const int& rhs) {
	*result = (lhs > rhs) ? lhs : rhs;
}

__device__ void Min(int * result, const int& lhs, const int& rhs) {
	*result = (lhs > rhs) ? rhs : lhs;
}

__device__ BinaryOperation binaryOperations[] = { Plus, Max, Min };

__global__ void ReduceByKernel(int* dst, int *src, int size, E_BOPER binaryOperationIndex) {
	__shared__ int sharedData[256];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	sharedData[tid] = (id < size) ? src[id] : 0;
	__syncthreads();
	for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			int tmp = sharedData[tid];
			if (tid + stride < size) {
				binaryOperations[binaryOperationIndex](sharedData + tid, sharedData[tid], sharedData[tid + stride]);
				printf("%2d: (%2d, %2d) => %2d\n", tid, tmp, sharedData[tid + stride], sharedData[tid]);
			}
		}
		__syncthreads();
		if (tid == 0) {
			printf("\n");
		}
	}
	if (tid == 0) {
		*dst = sharedData[0];
	}
}

int Reduce(int * ptr, int size, E_BOPER index) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);
	ReduceByKernel << <1, 32 >> > (deviceDst, deviceSrc, size, index);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);
	return hostDst;
}