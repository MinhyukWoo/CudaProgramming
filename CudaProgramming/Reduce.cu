#include "hip/hip_runtime.h"
#include"Reduce.hpp"
#include <hiprand/hiprand_kernel.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include<hip/hip_cooperative_groups.h>
#include<cooperative_groups/reduce.h>

__global__ void BadSum(int* dst, int *src, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		int prevDst = *dst;
		*dst += src[tid];
		printf("%2d: %2d + %2d = %2d\n", tid, src[tid], prevDst, *dst);
	}
}



int GetBadSum(int * ptr, int size) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);

	BadSum << <1, 256 >> > (deviceDst, deviceSrc, size);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);

	return hostDst;
}

typedef void (*BinaryOperation)(int *, const int&, const int&);

__device__ void Plus(int * result, const int& lhs, const int& rhs) {
	*result = lhs + rhs;
}

__device__ void Max(int * result, const int& lhs, const int& rhs) {
	*result = (lhs > rhs) ? lhs : rhs;
}

__device__ void Min(int * result, const int& lhs, const int& rhs) {
	*result = (lhs > rhs) ? rhs : lhs;
}

__device__ BinaryOperation binaryOperations[] = { Plus, Max, Min };

__global__ void ReduceByKernel(int* dst, int *src, int size, E_BOPER binaryOperationIndex) {
	__shared__ int sharedData[256];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	sharedData[tid] = (id < size) ? src[id] : 0;
	__syncthreads();
	for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			int tmp = sharedData[tid];
			if (tid + stride < size) {
				binaryOperations[binaryOperationIndex](sharedData + tid, sharedData[tid], sharedData[tid + stride]);
				printf("%2d: (%2d, %2d) => %2d\n", tid, tmp, sharedData[tid + stride], sharedData[tid]);
			}
		}
		__syncthreads();
		if (tid == 0) {
			printf("\n");
		}
	}
	if (tid == 0) {
		*dst = sharedData[0];
	}
}

int Reduce(int * ptr, int size, E_BOPER index) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);
	ReduceByKernel << <1, 32 >> > (deviceDst, deviceSrc, size, index);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);
	return hostDst;
}

__global__ void GetGLCM(
	unsigned int * comatrix, size_t comatrixColLength, WORD * image, size_t rowLength, size_t colLength, int deltaRow, int deltaCol
) {
	size_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
	size_t colIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if (!(rowIndex < rowLength && colIndex < colLength)){
		return;
	}
	if ((deltaRow > 0 && rowIndex + deltaRow >= rowLength)
		|| (deltaRow < 0 && rowIndex - deltaRow < 0)){
		return;
	}
	if ((deltaCol > 0 && colIndex + deltaCol >= colLength)
		|| (deltaCol < 0 && colIndex - deltaCol < 0)) {
		return;
	}

	size_t currIndex = rowIndex * colLength + colIndex;
	size_t nearIndex = (rowIndex + deltaRow) * colLength + (colIndex + deltaCol);
	size_t comatrixIndex = image[currIndex] * comatrixColLength + image[nearIndex];
	printf("%d, %d: %d, %d (%d)\n", rowIndex, colIndex, image[currIndex], image[nearIndex], comatrixIndex);

	atomicAdd(comatrix + comatrixIndex, (unsigned int)1);
}

unsigned int * GetComatrix(WORD * image, size_t rowLength, size_t colLength)
{
	size_t comtrixColLength = 9;
	size_t comatrixLength = comtrixColLength * comtrixColLength;

	unsigned int *hostComatrix = new unsigned int[comatrixLength] {0};
	unsigned int *deviceComatrix;
	hipMalloc(&deviceComatrix, sizeof(unsigned int) * comatrixLength);
	hipMemcpy(deviceComatrix, hostComatrix, sizeof(unsigned int) * comatrixLength, hipMemcpyHostToDevice);
	WORD *deviceImage;
	hipMalloc(&deviceImage, sizeof(WORD) * rowLength * colLength);
	hipMemcpy(deviceImage, image, sizeof(WORD) * rowLength * colLength, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks(1 + rowLength / 16, 1 + colLength / 16);
	GetGLCM << <blocks, threads >> > (deviceComatrix, comtrixColLength, deviceImage, rowLength, colLength, 0, 1);

	hipMemcpy(hostComatrix, deviceComatrix, sizeof(unsigned int) * comatrixLength, hipMemcpyDeviceToHost);

	hipFree(deviceComatrix);
	hipFree(deviceImage);
	return hostComatrix;
}
