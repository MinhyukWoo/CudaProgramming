#include "hip/hip_runtime.h"
#include"test.hpp"
#include <hiprand/hiprand_kernel.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include"hip/hip_runtime.h"
#include""

__global__ void BadSum(int* dst, int *src, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size) {
		int prevDst = *dst;
		*dst += src[tid];
		printf("%2d: %2d + %2d = %2d\n", tid, src[tid], prevDst, *dst);
	}
}



int GetBadSum(int * ptr, int size) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);

	BadSum << <1, 256 >> > (deviceDst, deviceSrc, size);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);

	return hostDst;
}

__global__ void SumReduced(int* dst, int *src, int size) {
	__shared__ int sharedData[256];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	sharedData[tid] = (id < size) ? src[id] : 0;
	__syncthreads();
	for (unsigned int stride = blockDim.x / 2; stride  > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			printf("%2d: %2d + %2d = %2d\n", tid, sharedData[tid], sharedData[tid + stride], sharedData[tid] + sharedData[tid + stride]);
			sharedData[tid] += sharedData[tid + stride];
		}
		__syncthreads();
		if (tid == 0) {
			printf("\n");
		}
	}
	if (tid == 0) {
		*dst = sharedData[0];
	}
}

int GetReducedSum(int * ptr, int size) {
	int* deviceSrc;
	hipMalloc(&deviceSrc, size * sizeof(int));
	hipMemcpy(deviceSrc, ptr, size * sizeof(int), hipMemcpyHostToDevice);

	int * deviceDst, hostTmp = 0;
	hipMalloc(&deviceDst, sizeof(int));
	hipMemcpy(deviceDst, &hostTmp, sizeof(int), hipMemcpyHostToDevice);

	SumReduced << <1, 16 >> > (deviceDst, deviceSrc, size);

	int hostDst;
	hipMemcpy(&hostDst, deviceDst, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceSrc);
	hipFree(deviceDst);
	return hostDst;
}